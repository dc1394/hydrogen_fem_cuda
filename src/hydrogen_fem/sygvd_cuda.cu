/*! \file sygvd_cuda.h
    \brief GPUで一般化固有値問題を解く関数の実装

    Copyright © 2021 @dc1394 All Rights Reserved.
    This software is released under the BSD 2-Clause License.    
*/

#include "sygvd_cuda.h"
#include <cassert>          // for assert 
#include <hipsolver.h>

namespace cuda {
    std::pair<std::vector<float>, std::vector<float> > sygvd_cuda(std::int32_t m, float const * A, float const * B)
    {
        hipsolverHandle_t cusolverH = nullptr;
        hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
        hipError_t cudaStat1 = hipSuccess;
        hipError_t cudaStat2 = hipSuccess;
        hipError_t cudaStat3 = hipSuccess;
        hipError_t cudaStat4 = hipSuccess;
        const int lda = m;

        std::vector<float> V(lda*m); // eigenvectors
        std::vector<float> W(m); // eigenvalues

        float *d_A = nullptr;
        float *d_B = nullptr;
        float *d_W = nullptr;
        int *devInfo = nullptr;
        float *d_work = nullptr;
        int  lwork = 0;
        int info_gpu = 0;
    
        // step 1: create cusolver/cublas handle
        cusolver_status = hipsolverDnCreate(&cusolverH);
        assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

        // step 2: copy A and B to device
        cudaStat1 = hipMalloc ((void**)&d_A, sizeof(float) * lda * m);
        cudaStat2 = hipMalloc ((void**)&d_B, sizeof(float) * lda * m);
        cudaStat3 = hipMalloc ((void**)&d_W, sizeof(float) * m);
        cudaStat4 = hipMalloc ((void**)&devInfo, sizeof(int));
        assert(hipSuccess == cudaStat1);
        assert(hipSuccess == cudaStat2);
        assert(hipSuccess == cudaStat3);
        assert(hipSuccess == cudaStat4);

        cudaStat1 = hipMemcpy(d_A, A, sizeof(float) * lda * m, hipMemcpyHostToDevice);
        cudaStat2 = hipMemcpy(d_B, B, sizeof(float) * lda * m, hipMemcpyHostToDevice);
        assert(hipSuccess == cudaStat1);
        assert(hipSuccess == cudaStat2);

        // step 3: query working space of sygvd
        hipsolverEigType_t itype = HIPSOLVER_EIG_TYPE_1; // A*x = (lambda)*B*x
        hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
        hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
        cusolver_status = hipsolverDnSsygvd_bufferSize(
            cusolverH,
            itype,
            jobz,
            uplo,
            m,
            d_A,
            lda,
            d_B,
            lda,
            d_W,
            &lwork);
        assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
        cudaStat1 = hipMalloc((void**)&d_work, sizeof(float)*lwork);
        assert(hipSuccess == cudaStat1);

        // step 4: compute spectrum of (A,B)
        cusolver_status = hipsolverDnSsygvd(
            cusolverH,
            itype,
            jobz,
            uplo,
            m,
            d_A,
            lda,
            d_B,
            lda,
            d_W,
            d_work,
            lwork,
            devInfo);
        cudaStat1 = hipDeviceSynchronize();
        assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
        assert(hipSuccess == cudaStat1);
    
        cudaStat1 = hipMemcpy(W.data(), d_W, sizeof(float)*m, hipMemcpyDeviceToHost);
        cudaStat2 = hipMemcpy(V.data(), d_A, sizeof(float)*lda*m, hipMemcpyDeviceToHost);
        cudaStat3 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
        assert(hipSuccess == cudaStat1);
        assert(hipSuccess == cudaStat2);
        assert(hipSuccess == cudaStat3);

        assert(0 == info_gpu);

        // free resources
        if (d_A    ) hipFree(d_A);
        if (d_B    ) hipFree(d_B);
        if (d_W    ) hipFree(d_W);
        if (devInfo) hipFree(devInfo);
        if (d_work ) hipFree(d_work);

        if (cusolverH) hipsolverDnDestroy(cusolverH);

        hipDeviceReset();

        return std::make_pair(std::move(W), std::move(V));
    }
}